#include "l1.cuh"


void saxpy(const int n, const float a, const float* x, const int incx, float* y, const int incy) {
  int block_size = 256;
  int grid_size = (n + block_size - 1) / block_size;

  float* d_x;
  float* d_y;

  // Allocate device memory
  hipMalloc((void**)&d_x, n * sizeof(float));
  hipMalloc((void**)&d_y, n * sizeof(float));

  // Copy data to device
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

  cuda_axpy<<<grid_size, block_size>>>(n, a, d_x, incx, d_y, incy);

  hipDeviceSynchronize();

  // Copy result back to host
  hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
}

void sscal(const int n, const float a, float* x, const int incx) {
  int block_size = 256;
  int grid_size = (n + block_size - 1) / block_size;

  float* d_x;

  // Allocate device memory
  hipMalloc((void**)&d_x, n * sizeof(float));

  // Copy data to device
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

  cuda_scal<<<grid_size, block_size>>>(n, a, d_x, incx);

  hipDeviceSynchronize();

  // Copy result back to host
  hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
}


float sasum(const float* h_x, int n) {
  const int BLOCK_SIZE = 256;  // Number of threads per block
  int num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

  // Allocate device memory
  float* d_x;
  float* d_partial_sums;
  hipMalloc(&d_x, n * sizeof(float));
  hipMalloc(&d_partial_sums, num_blocks * sizeof(float));

  // Copy data to device
  hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);

  cuda_asum<<<num_blocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(d_x, d_partial_sums, n);

  // Copy partial sums back to host
  float* h_partial_sums = new float[num_blocks];
  hipMemcpy(h_partial_sums, d_partial_sums, num_blocks * sizeof(float), hipMemcpyDeviceToHost);

  // Final reduction on the host
  float result = 0.0f;
  for (int i = 0; i < num_blocks; ++i) {
    result += h_partial_sums[i];
  }

  delete[] h_partial_sums;
  hipFree(d_x);
  hipFree(d_partial_sums);

  return result;
}
